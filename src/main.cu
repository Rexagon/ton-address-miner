#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <bip39.cuh>

#include "sha256.cuh"

#define MIN(a, b) ((a) < (b) ? (a) : (b))

constexpr auto ENTROPY_OFFSET = 8u;
constexpr auto MAX_BIP39_WORD_LEN = 8u;

constexpr auto THREAD_COUNT = 8192u;
constexpr auto THREADS_PER_BLOCK = 256u;

enum MnemonicType : uint32_t {
  Words12 = (128u << ENTROPY_OFFSET) | 4u,
  Words24 = (256u << ENTROPY_OFFSET) | 8u,
};

constexpr auto entropy_bits(MnemonicType type) -> size_t {
  return type >> ENTROPY_OFFSET;
}

constexpr auto checksum_bits(MnemonicType type) -> size_t {
  return type & 0xffu;
}

constexpr auto total_bits(MnemonicType type) -> size_t {
  return entropy_bits(type) + checksum_bits(type);
}

constexpr auto word_count(MnemonicType type) -> size_t {
  return total_bits(type) / 11u;
}

constexpr auto DEFAULT_MNEMONIC = MnemonicType::Words12;
constexpr auto MAX_PHRASE_LENGTH = (MAX_BIP39_WORD_LEN + 1) * word_count(DEFAULT_MNEMONIC);
constexpr auto ENTROPY_SIZE = entropy_bits(DEFAULT_MNEMONIC) >> 3u;
constexpr auto WORD_COUNT = word_count(DEFAULT_MNEMONIC);

#define MAKE_ONES(T, count) static_cast<T>((T{0b1u} << count) - T{0b1u})

inline void gpu_assert(hipError_t code, const char* file, int line, bool abort = true) {
  if (code == hipSuccess) {
    return;
  }
  fprintf(stderr, "GPU Assert: %s %s %d\n", hipGetErrorString(code), file, line);
  if (abort) {
    exit(code);
  }
}

#define UNWRAP_GPU(result) \
  { gpu_assert((result), __FILE__, __LINE__); }

inline void curand_assert(hiprandStatus_t code, const char* file, int line, bool abort = true) {
  if (code == hiprandStatus::HIPRAND_STATUS_SUCCESS) {
    return;
  }
  fprintf(stderr, "cuRAND Assert: STATUS(%d) %s %d\n", code, file, line);
  if (abort) {
    exit(code);
  }
}

#define UNWRAP_CURAND(result) \
  { curand_assert((result), __FILE__, __LINE__); }

__global__ void init_kernel(hiprandState* states, uint64_t seed) {
  const auto idx = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void generate_words(hiprandState* states, char* output) {
  const auto idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Generate entropy
  uint8_t buffer[ENTROPY_SIZE + 1];
  static_assert(ENTROPY_SIZE % 4 == 0, "Unaligned entropy used");

  for (auto i = 0; i < ENTROPY_SIZE / 4; ++i) {
    const auto randChunk = hiprand(&states[idx]);
    static_assert(sizeof(randChunk) == 4, "Invalid rand chunk length");

    for (auto j = 0; j < 4; ++j) {
      buffer[i * 4 + j] = reinterpret_cast<const uint8_t*>(&randChunk)[j];
    }
  }

  // Calculate checksum
  Sha256Context ctx{};
  sha256_init(&ctx);
  sha256_update(&ctx, buffer, ENTROPY_SIZE);
  buffer[ENTROPY_SIZE] = ctx.state[0] >> 24u;

  // Generate phrase
  auto phraseOffset = idx * MAX_PHRASE_LENGTH;
  auto bitsOffset = 0;
  for (auto i = 0; i < WORD_COUNT; ++i) {
    const auto j = bitsOffset / 8u;

    const auto first_byte_length = static_cast<uint16_t>(8u - (bitsOffset & 0b111u));

    const auto second_byte_length = MIN(11u - first_byte_length, 8u);
    const auto second_byte_offset = static_cast<uint16_t>(8u - second_byte_length);

    const auto third_byte_length = 11u - first_byte_length - second_byte_length;
    const auto third_byte_offset = static_cast<uint16_t>(8u - third_byte_length);

    uint16_t word_i{};
    word_i |= static_cast<uint16_t>(buffer[j] & MAKE_ONES(uint16_t, first_byte_length));
    word_i <<= second_byte_length;
    word_i |= static_cast<uint16_t>(buffer[j + 1] >> second_byte_offset);
    if (third_byte_length > 0) {
      word_i <<= third_byte_length;
      word_i |= static_cast<uint16_t>(buffer[j + 2] >> third_byte_offset);
    }

    bitsOffset += 11u;

    if (word_i >= 2048) {
      output[phraseOffset++] = 'A';
      continue;
    }

    if (i != 0) {
      output[phraseOffset++] = ' ';
    }
    auto word = reinterpret_cast<const char*>(&BIP39[word_i * 2]);
    for (auto k = 0; k < MAX_BIP39_WORD_LEN && word[k] != 0; ++k) {
      output[phraseOffset++] = word[k];
    }
  }
}

auto main() -> int {
  hiprandState* states;
  UNWRAP_GPU(hipMalloc(&states, THREAD_COUNT * sizeof(hiprandState)))

  char* output;
  UNWRAP_GPU(hipMallocManaged(&output, THREAD_COUNT * MAX_PHRASE_LENGTH))

  std::cout << "Word count: " << WORD_COUNT << std::endl;

  init_kernel<<<(THREAD_COUNT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(states, 1);
  UNWRAP_GPU(hipPeekAtLastError())
  UNWRAP_GPU(hipDeviceSynchronize())

  generate_words<<<(THREAD_COUNT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(states, output);
  UNWRAP_GPU(hipPeekAtLastError())
  UNWRAP_GPU(hipDeviceSynchronize())

  for (auto i = 0; i < THREAD_COUNT; ++i) {
    printf("%s\n", &output[i * MAX_PHRASE_LENGTH]);
  }

  UNWRAP_GPU(hipFree(states))
  UNWRAP_GPU(hipFree(output))
  return 0;
}
